
#include <hip/hip_runtime.h>
#include <cstdio>
#include <array>

template <size_t N>
__global__ void printCumSum(std::array<int, N> arr) {
    printf("Cumsum 0: %d\n", arr[0]);
    for (size_t i = 1; i < arr.size(); ++i) {
        arr[i] += arr[i - 1];
        printf("Cumsum %llu: %d\n", i, arr[i]);
    }
}

int main() {
    std::array<int, 4> arr{10, 5, 20, 23};
    printCumSum<<<1, 1>>>(arr);
    hipDeviceSynchronize();

    return 0;
}
